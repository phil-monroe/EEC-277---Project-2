#include "hip/hip_runtime.h"
def _BRANCHES_V_PERF_
#define _BRANCHES_V_PERF_

// includes -------------------------------------------------------------------
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "kernel.cu"	// Kernel to Maximize FLOPS

// Defines --------------------------------------------------------------------
// Hardware Dependent - NV GeForce 9500 GT

#define NUM_THREADS_PER_BLOCK 384	//	Taken from CUDA Occupancy Calc to maximize occupancy
#define NUM_ITERATIONS			16
#define NUM_BRANCHES_PER_THREAD 1

// Forward Declarations --------------------------------------------------------
void init_counters(float** h_counters, float** d_counters, unsigned int num_counters);
float runTest(int num_blocks);

// Main -----------------------------------------------------------------------
int main( int argc, char** argv) {
	printf("Testing Number of Branches vs. Performance");
	printf("Number of Threads/Blocks: %4d\n", NUM_THREADS_PER_BLOCK);
	printf("\n");
	
	FILE *file; 
	file = fopen("out.csv","a+");
	
	for(int iter = 0; iter < NUM_ITERATIONS; ++iter){
		printf("Iteration %d\n", iter);
		int num_branches = (int) pow(2.0f, (float) iter); 	// number of branches to run is 2^iter
		float perf = runTest(num_branches); 
		fprintf(file, "%d, %d, %f\n", iter, num_branches, perf);
	}
	
	fclose(file);
	exit(0);
}

// runTest --------------------------------------------------------------------
//		Runs a simple test to determine the FLOPS computed for a given
//		number of blocks
//
float runTest( int num_branches) {
	
	printf("Testing %4d Branches\n", num_branches);
	int num_threads = NUM_BLOCKS * NUM_THREADS_PER_BLOCK;
	int branch_gran = 32/num_branches;
	

	// Initialize counters on host and device to 0.0f
	float *h_counters, *d_counters;
	init_counters(&h_counters, &d_counters, num_threads);


	// Create and Start Timer
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// Run the test
	branch_perf_kernel<<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_counters, num_branches);
	hipDeviceSynchronize(); // Make sure all GPU computations are done
	
	
	// Record end time
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	float time_s = time/1000.0f;
	printf("Finished Test in %f s\n", time_s);

	// Check for errors
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
		printf("Error: %s\n", hipGetErrorString( error ));
		
	// Check array
	//hipMemcpy(h_counters, d_counters, threads * sizeof(float), hipMemcpyDeviceToHost);
	
	// for(int i = 0; i < threads; ++i){
	// 	printf("Thread %d: %f\n", i, h_counters[i]);
	// }

	// Calculate Performance
	float perf = NUM_FLOPS_PER_BLOCK*NUM_BLOCKS/(time_s* 1000.0f);
	printf("Total Perf: %.3f FLOPS\n", perf);
	printf("\n");
	
	// Cleanup
	free(h_counters);
	hipFree(d_counters);
	
	return perf;
}

// init_counters --------------------------------------------------------------
//		Initializes an array of floats that will be used to count FLOPS.
//
void init_counters(float** h_counters, float** d_counters, unsigned int num_counters){
	*h_counters = (float*) malloc( num_counters * sizeof(float));   // Allocate counters on host
	hipMalloc((void **) d_counters, num_counters*sizeof(float));   // Allocate counters on device

	// Initialize host counters to 0.0 ...
	for( unsigned int i = 0; i < num_counters; ++i)
		(*h_counters)[i] = 0.0f;
	// ... and copy to device
	hipMemcpy(*d_counters, *h_counters, num_counters * sizeof(float), hipMemcpyHostToDevice);
}


#endif /* _BLOCKS_V_PERF_ */
