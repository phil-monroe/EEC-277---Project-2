#ifndef _MAX_FLOPS_
#define _MAX_FLOPS_

// includes -------------------------------------------------------------------
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "max_flops_kernel.cu"	// Kernel to Maximize FLOPS

// Defines --------------------------------------------------------------------
#define NUM_BLOCKS 2048
#define NUM_THREADS_PER_BLOCK 384	//	Taken from CUDA Occupancy Calc to maximize occupancy

// Forward Declarations --------------------------------------------------------
void runTest( int argc, char** argv);
void init_counters(float** h_counters, float** d_counters, unsigned int num_counters);

// Main -----------------------------------------------------------------------
int main( int argc, char** argv) {
	runTest( argc, argv);

	exit(0);
}

// runTest --------------------------------------------------------------------
//		Runs a simple test to maximize the number of FLOPS computed on the GPU.
//
void runTest( int argc, char** argv) {

	// Hardware Dependent - NV GeForce 9500 GT
	unsigned int threads = NUM_BLOCKS * NUM_THREADS_PER_BLOCK;	
	
	printf("Number of Blocks:         %4d\n", NUM_BLOCKS);
	printf("Number of Threads/Blocks: %4d\n", NUM_THREADS_PER_BLOCK);
	printf("Number of Total Threads:  %4d\n", threads);
	printf("\n");
	

	// Initialize counters on host and device to 0.0f
	printf("Init counters\n");
	float *h_counters, *d_counters;
	init_counters(&h_counters, &d_counters, threads);


	// Create and Start Timer
	printf("Starting Test\n");
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	// Run the test
	int it = 10;
	for(int i = 0; i < it; i++)
		max_flops_kernel<<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_counters);
	hipDeviceSynchronize(); // Make sure all GPU computations are done
	
	
	// Record end time
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	float time_s = time/1000.0f;
	printf("Finished Test in %f s\n", time_s);

	// Check for errors
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
		printf("Error: %s\n", hipGetErrorString( error ));
		
	// Check array
	hipMemcpy(h_counters, d_counters, threads * sizeof(float), hipMemcpyDeviceToHost);
	
	// for(int i = 0; i < threads; ++i){
	// 	printf("Thread %d: %f\n", i, h_counters[i]);
	// }

	// Calculate GFLOPS
	unsigned long long total_flops = N_FLOPS_PER_KERNEL * threads * it;
	printf("Total FLOPs: %lld\n", total_flops);
	float gflops = total_flops/(time_s*1000000000.0f);
	printf("GFLOPS: %.3f\n", gflops);


	// Cleanup
	free(h_counters);
	hipFree(d_counters);
}

// init_counters --------------------------------------------------------------
//		Initializes an array of floats that will be used to count FLOPS.
//
void init_counters(float** h_counters, float** d_counters, unsigned int num_counters){
	*h_counters = (float*) malloc( num_counters * sizeof(float));   // Allocate counters on host
	hipMalloc((void **) d_counters, num_counters*sizeof(float));   // Allocate counters on device

	// Initialize host counters to 0.0 ...
	for( unsigned int i = 0; i < num_counters; ++i)
		(*h_counters)[i] = 0.0f;
	// ... and copy to device
	hipMemcpy(*d_counters, *h_counters, num_counters * sizeof(float), hipMemcpyHostToDevice);
}


#endif /* MAX_FLOPS */
